
#include <hip/hip_runtime.h>

__global__ void countDistances(int dimensions, float *teachingCollection, int teachingCollectionCount, float *classifyCollection, int classifyCollectionCount, float *distances)
{
	//compute :P
}

void cuda_knn(int dimensions, float *h_teachingCollection, int *h_teachedClasses, int teachingCollectionCount, float *h_classifyCollection, int *h_classifiedClasses, int classifyCollectionCount)
{
	int ierr;
	float *d_teachingCollection, *d_classifyCollection;
	ierr = hipMalloc(&d_teachingCollection, teachingCollectionCount*dimensions*sizeof(float));
	ierr = hipMalloc(&d_classifyCollection, classifyCollectionCount*dimensions*sizeof(float));

	ierr = hipMemcpy(d_teachingCollection, h_teachingCollection, teachingCollectionCount*dimensions, hipMemcpyDeviceToHost);
	ierr = hipMemcpy(d_classifyCollection, h_classifyCollection, classifyCollectionCount*dimensions, hipMemcpyDeviceToHost);

	float *d_distances;
	hipMalloc(&d_distances, teachingCollectionCount*classifyCollectionCount*sizeof(float));
	
	const int threadsPerBlock = 100;
	int blocksPerGrid = classifyCollectionCount / 100 + 1;
		
	countDistances<<<blocksPerGrid, threadsPerBlock>>>(dimensions, d_teachingCollection, teachingCollectionCount, d_classifyCollection, classifyCollectionCount, d_distances);
	hipFree(d_teachingCollection);
	hipFree(d_classifyCollection);
}
