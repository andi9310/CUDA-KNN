#include "hip/hip_runtime.h"
//#define DEBUG
//#define DEBUG_KERNEL

#include <iostream>

#include "utils.h"
#include "datatypes.h"

#ifdef DEBUG
	#include <cstdio>
#endif

// Prints any CUDA errors to stderr
// input:
// code - return code from CUDA function
void cudaCheckErrors(int code)
{
	if(code)
		std::cerr << "CUDA error no.: " << code << "\n";
}

int getNumberOfGpus()
{
	int n;
	cudaCheckErrors(hipGetDeviceCount(&n));
	return n;
}

void getGpusProperties(GpuProperties *properties)
{
	int n = getNumberOfGpus();
	for (int i=0; i<n; i++)
	{
		hipDeviceProp_t prop;
		cudaCheckErrors(hipGetDeviceProperties(&prop, i));
		
		properties[i].memory = prop.totalGlobalMem;
		properties[i].multiprocessors = prop.multiProcessorCount;
	}
}


void printCudaMem()
{
	    size_t free_byte ;

        size_t total_byte ;

        cudaCheckErrors(hipMemGetInfo( &free_byte, &total_byte ));
        double free_db = (double)free_byte ;

        double total_db = (double)total_byte ;

        double used_db = total_db - free_db ;

        std::cout << "GPU memory usage: used = " << used_db/1024.0/1024.0 << ", free = " << free_db/1024.0/1024.0 << " MB, total = " << total_db/1024.0/1024.0 << " MB\n";
}

// Finds slots for numbers in given array (ascending) and puts them there (sorting)
// input:
// myDistances - pointer to an array of distances, which will be fitted into myNearestDistances array
// K - size of myNearestDistances and myNearestIndexes arrays
// i - size of myDistances array
// output:
// myNearestIndexes - array of indices from myDistances array, wich were chosen as the lowest
__device__ int findSlot(float *myDistances, float *myNearestDistances, int *myNearestIndexes, int K, int i)
{
	int j;
	for (j=0; j<i; j++)
	{
		if (myDistances[i]<myNearestDistances[j]) //terrible nesting here
		{
			for (int k=K-1; k>j; k--)
			{
				myNearestDistances[k]=myNearestDistances[k-1];
				myNearestIndexes[k]=myNearestIndexes[k-1];
			}
			myNearestDistances[j]=myDistances[i];
			myNearestIndexes[j]=i;
			break;
		}
	}
	return j;
}

// Counts distances between point from classifyCollection to points from teachingCollection
// input:
// dimensions - dimensionality of space
// teachingCollection - pointer to an array, which represents points coordinates - each "dimensions" of elements represents one point (like vector) - teaching collection
// teachingCollectionCount - number of points in arrays mentioned above
// classifyCollection - pointer to an array, which represents points coordinates - each "dimensions" of elements represents one point (like vector) - classify collection
// classifyCollectionCount - number of points in array mentioned above
// output:
// distances - pointer to an array, which will be populated with distances to every point
__global__ void countDistances(int dimensions, float *teachingCollection, int teachingCollectionCount, float *classifyCollection, int classifyCollectionCount, float *distances, int* nearestIndexes, float* nearestDistances, int K, int *classCounters, int *teachedClasses, int *result)
{
	int tId = blockIdx.x*blockDim.x+threadIdx.x;
	int pointId = tId*dimensions;	
		
	if(tId >= classifyCollectionCount)
		return;
		
	#ifdef DEBUG_KERNEL
		printf("%d watek %d %d\n", tId, classifyCollectionCount, teachingCollectionCount);
	#endif
	
	for(int i = 0; i < teachingCollectionCount; ++i)
	{
		float distance = 0.0f;
		for(int j = 0; j < dimensions; ++j)
		{
			distance += (classifyCollection[pointId+j]-teachingCollection[i*dimensions+j])*(classifyCollection[pointId+j]-teachingCollection[i*dimensions+j]);
		}
		
		#ifdef DEBUG_KERNEL
			printf("%f\n", distance);
		#endif
		
		distances[teachingCollectionCount*tId+i] = distance;
	}
	
	int *myNearestIndexes = nearestIndexes+K*tId;
	float *myNearestDistances = nearestDistances+K*tId;
	float *myDistances = distances + tId * teachingCollectionCount;
	
	for (int i=0; i<K; i++)
	{
		int j = findSlot(myDistances, myNearestDistances, myNearestIndexes, K, i);
		
		if (j==i)
		{
			myNearestDistances[j]=myDistances[i];
			myNearestIndexes[j]=i;
		}
	}
	for (int i=K; i<teachingCollectionCount;i++)
	{
		findSlot(myDistances, myNearestDistances, myNearestIndexes, K, i);
	}
	
	for(int i = 0; i < K; ++i)
	{
		classCounters[tId*MAX_CLASS_NUMBER+teachedClasses[nearestIndexes[i+pointId]]]++;
	}

	int maxIndex = 0, maxValue = classCounters[tId*MAX_CLASS_NUMBER];
	for(int i = 1; i < MAX_CLASS_NUMBER; ++i)
	{
		if(classCounters[tId*MAX_CLASS_NUMBER+i] > maxValue)
		{
			maxIndex = i;
			maxValue = classCounters[tId*MAX_CLASS_NUMBER+i];
		}
	}
	result[tId] = maxIndex;
}

// Selects N-nearest points to a point from distances array
// input:
// K - number of nearest points to get
// distances - pointer to an array, which is populated with distances to every point
// teachingCollectionCount - number of points in teaching collection
// classifyCollectionCount - number of points in classify collection
// output:
// nearestIndexes - array of indices from distances array, wich were chosen as the lowest
__global__ void selectN(int K, float *distances, int teachingCollectionCount, int classifyCollectionCount, int *nearestIndexes, float *nearestDistances)
{		
	int myRank = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(myRank >= classifyCollectionCount)
		return;
		
	
	
}

// Selects N-nearest points to a point from distances array
// input:
// K - number of nearest points to get
// nearestIndexes - array of indices from distances array, wich were chosen as the lowest
// distances - pointer to an array, which is populated with distances to every point
// classifyCollectionCount - number of points in classify collection
// teachedClasses - pointer to an array, which represents classes of each point from teaching collection
// teachingCollectionCount - number of points in teaching collection
// output:
// result - pointer to an array, which will be populated with numbers of class each point was fitted into
__global__ void chooseClass(int K, int *nearestIndexes, int classifyCollectionCount, int *teachedClasses, int teachingCollectionCount, int *classCounters, int *result)
{
	int tId = blockIdx.x*blockDim.x+threadIdx.x;
	int pointId = tId*K;	
		
	if(tId >= classifyCollectionCount)
		return;

	
}

void cuda_knn(int K, int dimensions, float *h_teachingCollection, int *h_teachedClasses, int teachingCollectionCount, float *h_classifyCollection, int *h_classifiedClasses, int classifyCollectionCount, int threadsPerBlock)
{
	int blocksPerGrid = classifyCollectionCount / threadsPerBlock + 1;
	
	// Memory allocation block
	float *d_teachingCollection, *d_classifyCollection, *d_nearestDistances, *d_distances;
	cudaCheckErrors(hipMalloc(&d_teachingCollection, teachingCollectionCount*dimensions*sizeof(float)));
	cudaCheckErrors(hipMalloc(&d_classifyCollection, classifyCollectionCount*dimensions*sizeof(float)));
	cudaCheckErrors(hipMalloc(&d_nearestDistances, classifyCollectionCount*K*sizeof(float)));
	cudaCheckErrors(hipMalloc(&d_distances, teachingCollectionCount*classifyCollectionCount*sizeof(float)));
	
	int *d_classCounters, *d_teachedClasses, *d_result, *d_nearestIndexes;
	cudaCheckErrors(hipMalloc(&d_teachedClasses, teachingCollectionCount*sizeof(int)));
	cudaCheckErrors(hipMalloc(&d_result, classifyCollectionCount*sizeof(int)));	
	cudaCheckErrors(hipMalloc(&d_nearestIndexes, classifyCollectionCount*K*sizeof(int)));
	cudaCheckErrors(hipMalloc(&d_classCounters, classifyCollectionCount*MAX_CLASS_NUMBER*sizeof(int)));
	cudaCheckErrors(hipMemset(d_classCounters, 0, classifyCollectionCount*MAX_CLASS_NUMBER*sizeof(int)));
	
	// Copying parameters for kernels
	cudaCheckErrors(hipMemcpy(d_teachingCollection, h_teachingCollection, teachingCollectionCount*dimensions*sizeof(float), hipMemcpyHostToDevice));
	cudaCheckErrors(hipMemcpy(d_classifyCollection, h_classifyCollection, classifyCollectionCount*dimensions*sizeof(float), hipMemcpyHostToDevice));
	cudaCheckErrors(hipMemcpy(d_teachedClasses, h_teachedClasses, teachingCollectionCount*sizeof(int), hipMemcpyHostToDevice));
	
	// Kernel launches
	countDistances<<<blocksPerGrid, threadsPerBlock>>>(dimensions, d_teachingCollection, teachingCollectionCount, d_classifyCollection, classifyCollectionCount, d_distances, d_nearestIndexes, d_nearestDistances, K, d_classCounters, d_teachedClasses, d_result);

	#ifdef DEBUG
		float *h_distances = new float[teachingCollectionCount*classifyCollectionCount];
		cudaCheckErrors(hipMemcpy(h_distances, d_distances, teachingCollectionCount*classifyCollectionCount*sizeof(float), hipMemcpyDeviceToHost)); // copy calculated distances back to host
		printf("\n\n");
		for(int i = 0; i < teachingCollectionCount*classifyCollectionCount; ++i)
		{
			printf("distance %f\n", h_distances[i]);
		}

		float *h_nearestDistances = new float[classifyCollectionCount*K];
		int *h_nearestIndexes = new int[classifyCollectionCount*K];
		cudaCheckErrors(hipMemcpy(h_nearestDistances, d_nearestDistances, classifyCollectionCount*K*sizeof(float), hipMemcpyDeviceToHost));	
		cudaCheckErrors(hipMemcpy(h_nearestIndexes, d_nearestIndexes, classifyCollectionCount*K*sizeof(int), hipMemcpyDeviceToHost));
		printf("\n\n");
		for(int i = 0; i < classifyCollectionCount*K; ++i)
		{
			printf("nearest %d %f\n", h_nearestIndexes[i], h_nearestDistances[i]);
		}
	#endif	

	// Copying result back to host memory
	cudaCheckErrors(hipMemcpy(h_classifiedClasses, d_result, classifyCollectionCount*sizeof(int), hipMemcpyDeviceToHost));
		
	#ifdef DEBUG
		int *h_classCounters = new int[classifyCollectionCount*MAX_CLASS_NUMBER];
		cudaCheckErrors(hipMemcpy(h_classCounters, d_classCounters, classifyCollectionCount*MAX_CLASS_NUMBER*sizeof(int), hipMemcpyDeviceToHost));
		for(int i = 0; i < classifyCollectionCount*MAX_CLASS_NUMBER; ++i)
		{
			printf("counter %d\n", h_classCounters[i]);
		}
	#endif

	// Freeing memory
	cudaCheckErrors(hipFree(d_nearestDistances));
	cudaCheckErrors(hipFree(d_nearestIndexes));
	cudaCheckErrors(hipFree(d_teachingCollection));
	cudaCheckErrors(hipFree(d_classifyCollection));
	cudaCheckErrors(hipFree(d_distances));
	cudaCheckErrors(hipFree(d_classCounters));	
	cudaCheckErrors(hipFree(d_teachedClasses));
	cudaCheckErrors(hipFree(d_result));
	
	#ifdef DEBUG
		delete[] h_distances;
		delete[] h_nearestDistances;
		delete[] h_nearestIndexes;	
		delete[] h_classCounters;
	#endif
}
